#include "hip/hip_runtime.h"
#include <bits/stdc++.h>


// 展开最后一个32个线程的迭代
__device__ void WarpSharedMemReduce(volatile float *smem, int tid) {
  float x = smem[tid];
  if (blockDim.x >= 64) {
    x += smem[tid + 32];
    __syncwarp();
    smem[tid] = x;
    __syncwarp();
  }
  x += smem[tid + 16];
  __syncwarp();
  smem[tid] = x;
  __syncwarp();
  x += smem[tid + 8];
  __syncwarp();
  smem[tid] = x;
  __syncwarp();
  x += smem[tid + 4];
  __syncwarp();
  smem[tid] = x;
  __syncwarp();
  x += smem[tid + 2];
  __syncwarp();
  smem[tid] = x;
  __syncwarp();
  x += smem[tid + 1];
  __syncwarp();
  smem[tid] = x;
  __syncwarp();
}
// Note: using blockSize as a template arg can benefit from NVCC compiler
// optimization, which is better than using blockDim.x that is known in runtime.
template <int blockSize> __global__ void reduce_v4(float *d_in, float *d_out) {
  __shared__ float smem[blockSize];

  int tid = threadIdx.x;
  int i = blockIdx.x * (blockSize * 2) + threadIdx.x;
  // load: 每个线程加载两个元素到shared mem对应位置
  smem[tid] = d_in[i] + d_in[i + blockSize];
  __syncthreads();

  // compute: reduce in shared mem
  for (int s = blockDim.x / 2; s > 32; s >>= 1) {
    if (tid < s) {
      smem[tid] += smem[tid + s];
    }
    __syncthreads();
  }

  // last warp拎出来单独作reduce
  if (tid < 32) {
    WarpSharedMemReduce(smem, tid);
  }
  // store: write back to global mem
  if (tid == 0) {
    d_out[blockIdx.x] = smem[0];
  }
}

bool CheckResult(float *out, float groudtruth, int n) {
  float res = 0;
  for (int i = 0; i < n; i++) {
    res += out[i];
  }
  // printf("%f", res);
  if (res != groudtruth) {
    return false;
  }
  return true;
}

int main() {
  float milliseconds = 0;
  // const int N = 32 * 1024 * 1024;
  const int N = 25600000;
  hipSetDevice(0);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  const int blockSize = 256;
  int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
  // int GridSize = 100000;
  float *a = (float *)malloc(N * sizeof(float));
  float *d_a;
  hipMalloc((void **)&d_a, N * sizeof(float));

  float *out = (float *)malloc((GridSize) * sizeof(float));
  float *d_out;
  hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

  for (int i = 0; i < N; i++) {
    a[i] = 1.0f;
  }

  float groudtruth = N * 1.0f;

  hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 Grid(GridSize);
  dim3 Block(blockSize / 2);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  reduce_v4<blockSize / 2><<<Grid, Block>>>(d_a, d_out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
  printf("allcated %d blocks, data counts are %d \n", GridSize, N);
  bool is_right = CheckResult(out, groudtruth, GridSize);
  if (is_right) {
    printf("the ans is right\n");
  } else {
    printf("the ans is wrong\n");
    for (int i = 0; i < GridSize; i++) {
      printf("resPerBlock : %lf ", out[i]);
    }
    printf("\n");
    printf("groudtruth is: %f \n", groudtruth);
  }
  printf("reduce_v4 latency = %f ms\n", milliseconds);

  hipFree(d_a);
  hipFree(d_out);
  free(a);
  free(out);
}
