#include "hip/hip_runtime.h"
#include <bits/stdc++.h>


// multi-block reduce two pass
// latency: 1.815ms
template <int blockSize> __device__ void BlockSharedMemReduce(float *smem) {
  if (blockSize >= 1024) {
    if (threadIdx.x < 512) {
      smem[threadIdx.x] += smem[threadIdx.x + 512];
    }
    __syncthreads();
  }
  if (blockSize >= 512) {
    if (threadIdx.x < 256) {
      smem[threadIdx.x] += smem[threadIdx.x + 256];
    }
    __syncthreads();
  }
  if (blockSize >= 256) {
    if (threadIdx.x < 128) {
      smem[threadIdx.x] += smem[threadIdx.x + 128];
    }
    __syncthreads();
  }
  if (blockSize >= 128) {
    if (threadIdx.x < 64) {
      smem[threadIdx.x] += smem[threadIdx.x + 64];
    }
    __syncthreads();
  }
  // the final warp
  if (threadIdx.x < 32) {
    volatile float *vshm = smem;
    if (blockDim.x >= 64) {
      vshm[threadIdx.x] += vshm[threadIdx.x + 32];
    }
    vshm[threadIdx.x] += vshm[threadIdx.x + 16];
    vshm[threadIdx.x] += vshm[threadIdx.x + 8];
    vshm[threadIdx.x] += vshm[threadIdx.x + 4];
    vshm[threadIdx.x] += vshm[threadIdx.x + 2];
    vshm[threadIdx.x] += vshm[threadIdx.x + 1];
  }
}

template <int blockSize>
__global__ void reduce_v6(float *d_in, float *d_out, int nums) {
  __shared__ float smem[blockSize];

  unsigned int tid = threadIdx.x;
  unsigned int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int total_thread_num = blockDim.x * gridDim.x;
  // unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
  // smem[tid] = d_in[i] + d_in[i + blockDim.x];
  // load: 每个线程负责若干个元素的thread local求和，最后存到shared mem对应位置
  float sum = 0.0f;
  for (int32_t i = gtid; i < nums; i += total_thread_num) {
    sum += d_in[i];
  }
  smem[tid] = sum;
  __syncthreads();
  // compute: reduce in shared mem
  BlockSharedMemReduce<blockSize>(smem);

  // store: write back to global mem
  if (tid == 0) {
    d_out[blockIdx.x] = smem[0];
  }
}

bool CheckResult(float *out, float groudtruth, int n) {
  if (*out != groudtruth) {
    return false;
  }
  return true;
}

int main() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int maxblocks = deviceProp.maxGridSize[0];
  const int blockSize = 256;
  const int N = 25600000;
  int gridSize = std::min((N + blockSize - 1) / blockSize, maxblocks);

  float milliseconds = 0;
  float *a = (float *)malloc(N * sizeof(float));
  float *d_a;
  hipMalloc((void **)&d_a, N * sizeof(float));

  float *out = (float *)malloc((gridSize) * sizeof(float));
  float *d_out;
  float *part_out; //新增part_out存储每个block reduce的结果
  hipMalloc((void **)&d_out, 1 * sizeof(float));
  hipMalloc((void **)&part_out, (gridSize) * sizeof(float));
  float groudtruth = N;

  for (int i = 0; i < N; i++) {
    a[i] = 1;
  }

  hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 Grid(gridSize);
  dim3 Block(blockSize);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  reduce_v6<blockSize><<<Grid, Block>>>(d_a, part_out, N);
  reduce_v6<blockSize><<<1, Block>>>(part_out, d_out, gridSize);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(out, d_out, 1 * sizeof(float), hipMemcpyDeviceToHost);
  bool is_right = CheckResult(out, groudtruth, 1);
  if (is_right) {
    printf("the ans is right\n");
  } else {
    printf("the ans is wrong\n");
    for (int i = 0; i < 1; i++) {
      printf("%lf ", out[i]);
    }
    printf("\n");
  }
  printf("reduce_v6 latency = %f ms\n", milliseconds);

  hipFree(d_a);
  hipFree(d_out);
  hipFree(part_out);
  free(a);
  free(out);
}
