#include "hip/hip_runtime.h"
#include <bits/stdc++.h>


// latency: 2.300ms
template <int blockSize> __global__ void reduce_v2(float *d_in, float *d_out) {
  __shared__ float smem[blockSize];

  unsigned int tid = threadIdx.x;
  unsigned int gtid = blockIdx.x * blockSize + threadIdx.x;
  // load: 每个线程加载一个元素到shared mem对应位置
  smem[tid] = d_in[gtid];
  __syncthreads();

  // compute: reduce in shared mem
  // 0,1 1,2 store in same bank
  // 所以最好是从远的更新
  for (unsigned int index = blockDim.x / 2; index > 0; index >>= 1) {
    if (tid < index) {
      smem[tid] += smem[tid + index];
    }
    __syncthreads();
  }

  // store: write back to global mem
  if (tid == 0) {
    d_out[blockIdx.x] = smem[0];
  }
}

bool CheckResult(float *out, float groudtruth, int n) {
  float res = 0;
  for (int i = 0; i < n; i++) {
    res += out[i];
  }
  if (res != groudtruth) {
    return false;
  }
  return true;
}

int main() {
  float milliseconds = 0;
  // const int N = 32 * 1024 * 1024;
  const int N = 25600000;
  hipSetDevice(0);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  const int blockSize = 256;
  int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
  // int GridSize = 100000;
  float *a = (float *)malloc(N * sizeof(float));
  float *d_a;
  hipMalloc((void **)&d_a, N * sizeof(float));

  float *out = (float *)malloc((GridSize) * sizeof(float));
  float *d_out;
  hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

  for (int i = 0; i < N; i++) {
    a[i] = 1.0f;
  }

  float groudtruth = N * 1.0f;

  hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 Grid(GridSize);
  dim3 Block(blockSize);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  reduce_v2<blockSize><<<Grid, Block>>>(d_a, d_out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
  printf("allcated %d blocks, data counts are %d", GridSize, N);
  bool is_right = CheckResult(out, groudtruth, GridSize);
  if (is_right) {
    printf("the ans is right\n");
  } else {
    printf("the ans is wrong\n");
    // for(int i = 0; i < GridSize;i++){
    // printf("res per block : %lf ",out[i]);
    //}
    // printf("\n");
    printf("groudtruth is: %f \n", groudtruth);
  }
  printf("reduce_v2 latency = %f ms\n", milliseconds);

  hipFree(d_a);
  hipFree(d_out);
  free(a);
  free(out);
}
