#include "hip/hip_runtime.h"
#include <algorithm>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>

#define DEBUG
#ifdef DEBUG
#define checkCudaErrors(func)                                                  \
  {                                                                            \
    hipError_t e = (func);                                                    \
    if (e != hipSuccess)                                                      \
      printf("%s %d CUDA: %s\n", __FILE__, __LINE__, hipGetErrorString(e));   \
  }
#else
#define checkCudaErrors(func) (func)
#endif
constexpr int MAX_DEPTH = 16;
constexpr int INSERTION_SORT = 32;

__device__ void selection_sort(float *data, int left, int right) {
  // printf("selection sort from %d to %d\n",left,right);
  for (int i = left; i < right; i++) {
    float min_val = data[i];
    int min_idx = i;
    for (int j = i + 1; j <= right; j++) {
      if (data[j] < min_val) {
        min_idx = j;
        min_val = data[j];
      }
    }
    if (i != min_idx) {
      data[min_idx] = data[i];
      data[i] = min_val;
    }
  }
}

__global__ void quick_sort(float *data, int left, int right, int depth) {
  if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT) {
    selection_sort(data, left, right);
    return;
  }
  // printf("DEPTH %d\n",depth);

  float *left_ptr = data + left;
  float *right_ptr = data + right;

  float pivot = data[right];

  // partion

  while (left_ptr <= right_ptr) {
    float left_val = *left_ptr;
    float right_val = *right_ptr;
    while (left_val < pivot) {
      left_ptr++;
      left_val = *left_ptr;
    }

    while (right_val > pivot) {
      right_ptr--;
      right_val = *right_ptr;
    }

    if (left_ptr <= right_ptr) {
      *left_ptr = right_val;
      left_ptr++;
      *right_ptr = left_val;
      right_ptr--;
    }
  }

  int n_right = right_ptr - data;
  int n_left = left_ptr - data;

  if (left < (right_ptr - data)) {
    hipStream_t l_stream;
    // 设置非阻塞流
    hipStreamCreateWithFlags(&l_stream, hipStreamNonBlocking);
    quick_sort<<<1, 1, 0, l_stream>>>(data, left, n_right, depth + 1);
  }

  if ((left_ptr - data) < right) {
    hipStream_t r_stream;
    // 设置非阻塞流
    hipStreamCreateWithFlags(&r_stream, hipStreamNonBlocking);
    quick_sort<<<1, 1, 0, r_stream>>>(data, n_left, right, depth + 1);
  }
}

void run_qsort(float *data, int nitems) {
  // Prepare CDP for the max depth 'MAX_DEPTH'.
  checkCudaErrors(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));
  printf("run own qsort\n");
  int left = 0;
  int right = nitems - 1;
  quick_sort<<<1, 1>>>(data, left, right, 0);
  checkCudaErrors(hipDeviceSynchronize());
}

int main() {
  float milliseconds;
  const int N = 5e4;

  float *c_data = (float *)malloc(N * sizeof(float));
  std::generate(c_data, c_data + N, rand);


  float *g_data = nullptr;

  checkCudaErrors(hipMalloc(&g_data, N * sizeof(float)));
  checkCudaErrors(
      hipMemcpy(g_data, c_data, N * sizeof(float), hipMemcpyHostToDevice));

  thrust::device_ptr<float> t_data(g_data);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  // thrust::sort( t_data, t_data + N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  float *t_result = (float *)malloc(N * sizeof(float));
  checkCudaErrors(
      hipMemcpy(t_result, g_data, N * sizeof(float), hipMemcpyDeviceToHost));
  printf("thrust sort latency = %f ms\n", milliseconds);

  checkCudaErrors(
      hipMemcpy(g_data, c_data, N * sizeof(float), hipMemcpyHostToDevice));

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  run_qsort(g_data, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("own sort latency = %f ms\n", milliseconds);


  float *g_result = (float *)malloc(N * sizeof(float));
  checkCudaErrors(
      hipMemcpy(g_result, g_data, N * sizeof(float), hipMemcpyDeviceToHost));

  for (int i = 0; i < N; i++) {
    if (g_result[i] != t_result[i]) {
      printf("%d-th error", i);
      return 0;
    }
  }

  checkCudaErrors(hipFree(g_data));
  free(c_data);
  free(g_result);
  return 0;
}
