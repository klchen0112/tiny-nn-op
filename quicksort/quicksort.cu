#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define DEBUG
#ifdef DEBUG
#define checkCudaErrors(func)                                                  \
  {                                                                            \
    hipError_t e = (func);                                                    \
    if (e != hipSuccess)                                                      \
      printf("%s %d CUDA: %s\n", __FILE__, __LINE__, hipGetErrorString(e));   \
  }
#else
#define checkCudaErrors(func) (func)
#endif
constexpr int MAX_DEPTH = 16;
constexpr int INSERTION_SORT = 32;

__device__ void selection_sort(float *data, int left, int right) {
  for (int i = left; i < right; i++) {
    float min_val = data[i];
    int min_idx = i;
    for (int j = i + 1; j <= right; j++) {
      if (data[j] < min_val) {
        min_idx = j;
        min_val = data[j];
      }
    }
    if (i != min_idx) {
      data[min_idx] = data[i];
      data[i] = min_val;
    }
  }
}

__global__ void quick_sort(float *data, int left, int right, int depth) {
  if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT) {
    selection_sort(data, left, right);
    return;
  }

  float *left_ptr = data + left;
  float *right_ptr = data + right;

  float pivot = data[right];

  // partion

  while (left_ptr <= right_ptr) {
    float left_val = *left_ptr;
    float right_val = *right_ptr;
    while (left_val < pivot) {
      left_ptr++;
      left_val = *left_ptr;
    }

    while (right_val > pivot) {
      right_ptr--;
      right_val = *right_ptr;
    }

    if (left_ptr < right_ptr) {
      *left_ptr = right_val;
      left_ptr++;
      *right_ptr = left_val;
      right_ptr--;
    }
  }

  int n_right = right_ptr - data;
  int n_left = left_ptr - data;

  if (left < (right_ptr - data)) {
    hipStream_t l_stream;
    // 设置非阻塞流
    hipStreamCreateWithFlags(&l_stream, hipStreamNonBlocking);
    quick_sort<<<1, 1, 0, l_stream>>>(data, left, n_right, depth + 1);
    hipStreamDestroy(l_stream);
  }

  if ((left_ptr - data) < right) {
    hipStream_t r_stream;
    // 设置非阻塞流
    hipStreamCreateWithFlags(&r_stream, hipStreamNonBlocking);
    quick_sort<<<1, 1, 0, r_stream>>>(data, n_left, right, depth + 1);
    hipStreamDestroy(r_stream);
  }
}

void run_qsort(float *data, int nitems) {
  // Prepare CDP for the max depth 'MAX_DEPTH'.
  GPU_CHECK(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));

  int left = 0;
  int right = nitems - 1;
  quick_sort<<<1, 1>>>(data, left, right, 0);
  GPU_CHECK(hipDeviceSynchronize());
}

int main() {
  float milliseconds;
  const int N = 1e7;

  float *c_data = (float *)malloc(N * sizeof(float));

  float *g_data;

  checkCudaErrors(hipMalloc(&g_data, N * sizeof(float)));
  checkCudaErrors(
      hipMemcpy(c_data, g_data, N * sizeof(float), hipMemcpyHostToDevice));

  sort(c_data, c_data + N);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  run_qsort(g_data, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("reduce_v0 latency = %f ms\n", milliseconds);

  float *g_result = (float *)malloc(N * sizeof(float));
  checkCudaErrors(
      hipMemcpy(g_data, g_result, N * sizeof(float), hipMemcpyDeviceToHost));

  for (int i = 0; i < N; i++) {
    if (g_result[i] != c_data[i]) {
      printf("%d-th error", i);
      return 0;
    }
  }

  printf("WarpSoftmax latency = %f ms\n", milliseconds);
  checkCudaErrors(hipFree(g_data));
  free(g_data);
  free(g_result);
  return 0;
}
